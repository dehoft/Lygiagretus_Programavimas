#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "iostream"
#include "string"
#include "fstream"
#include <thread>
#include <sstream>
#include <iomanip>
#include <algorithm>
#include <mutex>

#include <stdio.h>

using namespace std;

#define DEBUG_MODE 1	// output to console
#define DETAILED_DEBUG 0	// output to console

#if DEBUG_MODE
#define DEBUG
#endif

#if DETAILED_DEBUG
#define DDEBUG
#endif

const char srcfile[] = "../DeksnysE_L4.txt";
const char rezfile[] = "../DeksnysE_L4_rez.txt";
const int datasize = 15;
//const int threadcount = 5;
char gsepLine[] = "+----";
char gnameLine[] = "|Gija";
int gcols = 4;
char sepLine[] = "+---+----+-------------------+--------------------+------------------+--------+\n";
char nameLine[] = "|Nr.| ID | Modelis           | Savininkas         | Pagaminimo metai | Svoris |\n";
int cols[7];

char sepLine2[] = "+---+----+---------------------------------------------------------+-----------------------------------------------------------------+------------------+--------+\n";
char nameLine2[] = "|Nr.| ID | Modelis                                                 | Savininkas                                                      | Pagaminimo metai | Svoris |\n";
int cols2[7];

// class for data to store
class car
{
public:
	char model[100];
	char owner[100];
	int year_made;
	double mass;
	int id;
	string ToString(int num, int col[]) {
		stringstream ss;

		ss << "|" << setw(col[0]) << num;
		ss << "|" << setw(col[1]) << id;
		ss << "|" << left << setw(col[2]);
		if (model != NULL) {
			ss << model;
		}
		else {
			ss << "NULL";
		}
		ss << "|" << left << setw(col[3]);
		if (owner != NULL) {
			ss << owner;
		}
		else {
			ss << "NULL";
		}
		ss << "|" << setw(col[4]) << year_made;
		ss << "|" << setw(col[5]) << mass << "|" << endl;
		string ans = ss.str();
		//ss >> ans;
		return ans;
	}

	car() {}

	car(char model[100], char owner[100], int met, double svo) {
		strcpy(model, model);
		strcpy(owner, owner);
		year_made = met;
		mass = svo;
	}

	static bool compare(car x, car y) {
		return x.year_made < y.year_made;
	}
};

// global variables
const int nrez = 100;
const int n = 12;
const int wt = 6; //writer threads count
const int rt = 6; //reader threads count


car P[wt][n];
car Result[n];
int writersizes[wt];
int readersizes[rt];
ofstream out(rezfile);

// functions 
void readData();
void writeData(char title[], car P[], int size);
void writeResult(char title[], car P[], int size);
void GetColsWidth();


#ifdef DEBUG
void DebugWriteData(char title[], car P[], int size);
void DebugWriteResult(char title[], car P[], int size);
#endif	

/// <summary>
/// reading data from data source file
/// </summary>
void readData() {
	ifstream src(srcfile);
	int size = 0;
	
	for (int i = 0; i < wt; i++) {
		src >> size;
		writersizes[i] = size;
	
		for (int j = 0; j < size; j++)
		{
			char v[100], p[100];

			src >> v >> p >> P[i][j].year_made >> P[i][j].mass; 
			strcpy(P[i][j].model, v);
			strcpy(P[i][j].owner, p);
			P[i][j].id = j + 1;
#ifdef DDEBUG
			cout << P[i][j].name << " " << P[i][j].surname << " " << P[i][j].birth_year << " " << P[i][j].mass << endl;
#endif
		}
		sort(P[i], P[i] + size, (new car)->compare);
	}
	src.close();
}

/// <summary>
/// writes data of array into tables to file 
/// </summary>
/// <param name="title"> title of table </param>
/// <param name="P"> array to print to table </param>
/// <param name="size"> size of data to print to table </param>
void writeData(char title[], car P[], int size) {
#ifdef DEBUG
	DebugWriteData(title, P, size);
#endif
	out << title << endl;
	out << sepLine << nameLine << sepLine;
	for (int i = 0; i < size; i++) {
		out << P[i].ToString(i + 1, cols);
	}
	out << sepLine << endl;
}

/// <summary>
/// writes data of array into tables to file 
/// </summary>
/// <param name="title"> title of table </param>
/// <param name="P"> array to print to table </param>
/// <param name="size"> size of data to print to table </param>
void writeResult(char title[], car P[], int size) {
#ifdef DEBUG
	DebugWriteResult(title, P, size);
#endif
	out << title << endl;
	out << sepLine2 << nameLine2 << sepLine2;
	for (int i = 0; i < size; i++) {
		out << P[i].ToString(i + 1, cols2);
	}
	out << sepLine2 << endl;
}

#ifdef DEBUG
/// <summary>
/// writes data of array into tables to console
/// </summary>
/// <param name="title"> title of table </param>
/// <param name="P"> array to print to table </param>
/// <param name="size"> size of data to print to table </param>
void DebugWriteData(char title[], car P[], int size) {
	cout << title << endl;
	cout << sepLine << nameLine << sepLine;
	for (int i = 0; i < size; i++) {
		cout << P[i].ToString(i + 1, cols);
	}
	cout << sepLine << endl;
}
#endif	

#ifdef DEBUG
/// <summary>
/// writes data of array into tables to console
/// </summary>
/// <param name="title"> title of table </param>
/// <param name="P"> array to print to table </param>
/// <param name="size"> size of data to print to table </param>
void DebugWriteResult(char title[], car P[], int size) {
	cout << title << endl;
	cout << sepLine2 << nameLine2 << sepLine2;
	for (int i = 0; i < size; i++) {
		cout << P[i].ToString(i + 1, cols2);
	}
	cout << sepLine2 << endl;
}
#endif	

/// <summary>
/// get width of table columns for dynamic printing
/// </summary>
void GetColsWidth() {
	int col = 0, size = 0;
	int len = sizeof(sepLine);
	for (int i = 1; i < len; i++) {
		if (sepLine[i] == '+') {
			cols[col] = size;
			size = 0;
			col++;
		}
		else {
			size++;
		}
	}
	col = 0, size = 0;
	len = sizeof(sepLine2);
	for (int i = 1; i < len; i++) {
		if (sepLine2[i] == '+') {
			cols2[col] = size;
			size = 0;
			col++;
		}
		else {
			size++;
		}
	}
}

//////////////////////////////CUDA///////////////////////////

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__device__ char * my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

__device__ char * my_strcat(char *dest, const char *src) {
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest + i, src);
	return dest;
}

__global__ void add(car** Pe, car* result) {
	int idx = threadIdx.x;
	result[idx].id = 0;
	result[idx].mass = 0;
	result[idx].year_made = 0;

	for (int i = 0; i < wt; i++)
	{
		result[idx].id += Pe[i][idx].id;
		char *name = result[idx].model;
		my_strcat(name, Pe[i][idx].model);
		char *surname = result[idx].owner;
		my_strcat(surname, Pe[i][idx].owner);
		result[idx].mass += Pe[i][idx].mass;
		result[idx].year_made += Pe[i][idx].year_made;
	}
}

int main()
{
	GetColsWidth();
	readData();
	for (int i = 0; i < wt; i++)
	{
		char pav[18] = "Duomenu rinkinys ";
		char integer_string[1];

		sprintf(integer_string, "%d", i + 1);
		strcat(pav, integer_string);

		writeData(pav, P[i], writersizes[i]);
	}

	//allocate memory on GPU
	int size = n * sizeof(car);
	car **Cuda_P;
	hipMalloc((void**)&Cuda_P, wt * sizeof(car*));
	//copy
	for (int i = 0; i < wt; i++)
	{
		car * Cuda_P_child;
		hipMalloc((void**)&Cuda_P_child, size);
		hipMemcpy(Cuda_P_child, P[i], size, hipMemcpyHostToDevice);
		hipMemcpy(&Cuda_P[i], &Cuda_P_child, sizeof(car*), hipMemcpyHostToDevice);
	}
	car * Cuda_Res;
	hipMalloc((void**)&Cuda_Res, size);

	// launching proc

	add << <1, n >> >(Cuda_P, Cuda_Res);

	hipDeviceSynchronize();

	//From GPU to CPU
	hipMemcpy(Result, Cuda_Res, size, hipMemcpyDeviceToHost);

	//free up gpu memory
	for (int i = 0; i < wt; i++)
	{
		hipFree(&Cuda_P[i]);
	}
	hipFree(Cuda_P);

	writeResult("Resultatas", Result, n);

#ifdef DEBUG
	cout << "program done";
	int a;
	cin >> a;
#endif
	out.close();

	return 0;
}
